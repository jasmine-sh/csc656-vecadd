#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<24; // 1M elements

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  
  // data transfer time workaround
  int deviceID=0;
  hipMemPrefetchAsync((void *)x, N*sizeof(float), deviceID);
  hipMemPrefetchAsync((void *)y, N*sizeof(float), deviceID);


  // Run kernel on 1M elements on the CPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  std::cout << "# THREAD BLOCKS: " << numBlocks << std::endl;
  add<<<numBlocks, blockSize>>>(N, x, y);


  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}